#include <hip/hip_runtime.h>
#include <iostream>
#include "spd_gpu_blas.hpp"

using namespace std;

 hipStream_t streams[4];

 int cur_stream = 0;
 bool initialized = false;

void init_spd()
{
  if (!initialized)
    {
      for (int i = 0; i < SPD_NUM_STREAMS; ++i){
	hipError_t err = hipStreamCreateWithFlags(&streams[i],hipStreamNonBlocking);
	if (err != hipSuccess)
	  cout<<"Err create stream"<<endl;
      }

      initialized = true;
    }
}


void finalize_spd()
{
  if (initialized)
    {
      for (int i = 0; i < SPD_NUM_STREAMS; ++i)
	hipStreamDestroy(streams[i]);
      
      hipDeviceReset();

      initialized = false;
    }
}

