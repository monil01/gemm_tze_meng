#include "hip/hip_runtime.h"
#include <iostream>
#include "spd_gpu_blas.hpp"

using namespace std;

#define CUDA_CHECK(routine, msg)   \
  { \
    hipError_t status;            \
    status = (routine);            \
    if (status != hipSuccess){    \
      cout<<msg<<endl;	           \
      return status; \
    } \
  }
template<typename T>
__global__ void _gemm_ker(size_t M, size_t N, size_t K,
			     T alpha,
			     T *A, size_t lda, T *B, size_t ldb,
			     T beta,
			     T *C, size_t ldc)
{
  if (blockIdx.x == 0 && threadIdx.x == 0 &&
      blockIdx.y == 0 && threadIdx.y == 0)
    {
      for (int i = 0; i != M; ++i)
	for (int j = 0; j != N; ++j)
	  for (int p = 0; p != K; ++p)
	    {
	      C[i*ldc + j] += A[i*lda + p] * B[p*ldb + j];
	    }
    }
}


template<typename T>
void spd_gpu_gemm(size_t M, size_t N, size_t K,
		  T alpha,
		  T *A, size_t lda, T *B, size_t ldb,
		  T beta,
		  T *C, size_t ldc)
{
  _gemm_ker<<<4, 128, 0>>>(M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);
}

auto spd_gpu_dgemm = spd_gpu_gemm<double>;
auto spd_gpu_sgemm = spd_gpu_gemm<float>;
